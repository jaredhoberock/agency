#include <agency/executor.hpp>
#include <agency/cuda/executor.hpp>
#include <iostream>

int main()
{
  using namespace agency::detail::executor_traits_detail;

  std::cout << "is_executor<agency::sequential_executor>: " << agency::is_executor<agency::sequential_executor>::value << std::endl;
  std::cout << "has_then_execute<agency::sequential_executor>: " << has_any_multi_agent_then_execute<agency::sequential_executor>::value << std::endl;

  std::cout << "is_executor<agency::concurrent_executor>: " << agency::is_executor<agency::concurrent_executor>::value << std::endl;
  std::cout << "has_then_execute<agency::concurrent_executor>: " << has_any_multi_agent_then_execute<agency::concurrent_executor>::value << std::endl;

  std::cout << "is_executor<agency::parallel_executor>: " << agency::is_executor<agency::parallel_executor>::value << std::endl;
  std::cout << "has_then_execute<agency::parallel_executor>: " << has_any_multi_agent_then_execute<agency::parallel_executor>::value << std::endl;

  std::cout << "is_executor<agency::scoped_executor<agency::concurrent_executor,agency::sequential_executor>>: " << agency::is_executor<agency::scoped_executor<agency::concurrent_executor,agency::sequential_executor>>::value << std::endl;
  std::cout << "has_then_execute<agency::scoped_executor<agency::concurrent_executor,agency::sequential_executor>>: " << has_any_multi_agent_then_execute<agency::scoped_executor<agency::concurrent_executor,agency::sequential_executor>>::value << std::endl;


  std::cout << "is_executor<grid_executor>: " << agency::is_executor<agency::cuda::grid_executor>::value << std::endl;
  std::cout << "has_then_execute<grid_executor>: " << has_any_multi_agent_then_execute<agency::cuda::grid_executor>::value << std::endl;

  std::cout << "is_executor<block_executor>: " << agency::is_executor<agency::cuda::block_executor>::value << std::endl;
  std::cout << "has_then_execute<block_executor>: " << has_any_multi_agent_then_execute<agency::cuda::block_executor>::value << std::endl;

  std::cout << "is_executor<cuda::parallel_executor>: " << agency::is_executor<agency::cuda::parallel_executor>::value << std::endl;
  std::cout << "has_then_execute<cuda::parallel_executor>: " << has_any_multi_agent_then_execute<agency::cuda::parallel_executor>::value << std::endl;

  std::cout << "is_executor<cuda::concurrent_executor>: " << agency::is_executor<agency::cuda::concurrent_executor>::value << std::endl;
  std::cout << "has_then_execute<cuda::concurrent_executor>: " << has_any_multi_agent_then_execute<agency::cuda::concurrent_executor>::value << std::endl;

  return 0;
}

