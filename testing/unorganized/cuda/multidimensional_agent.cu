#include <iostream>
#include <agency/agency.hpp>
#include <agency/cuda.hpp>

const agency::basic_execution_policy<agency::parallel_agent_2d, agency::cuda::parallel_executor> par2d{};

struct functor
{
  __device__
  void operator()(agency::parallel_agent_2d& self)
  {
    printf("Hello world from agent {%d, %d}\n", agency::get<0>(self.index()), agency::get<1>(self.index()));
  }
};

int main()
{
  auto exec = par2d({0,0}, {2,2});

  agency::bulk_invoke(exec, functor());

  hipError_t error = hipDeviceSynchronize();

  std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;

  return 0;
}

