#include "hip/hip_runtime.h"
#include <agency/cuda/executor/grid_executor.hpp>
#include <agency/bulk_invoke.hpp>

struct hello_world
{
  __device__
  void operator()(agency::cuda::grid_executor_2d::index_type index)
  {
    auto outer = agency::detail::get<0>(index);
    auto inner = agency::detail::get<1>(index);
    printf("Hello world from block {%d,%d}, thread {%d,%d}\n", outer[0], outer[1], inner[0], inner[1]);
  }
};

int main()
{
  agency::cuda::grid_executor_2d ex;

  auto num_blocks = agency::uint2{2,2};
  auto num_threads = agency::uint2{2,2};
  agency::cuda::grid_executor_2d::shape_type shape = {num_blocks, num_threads};

  std::cout << "Testing execute on host" << std::endl;
  agency::executor_traits<agency::cuda::grid_executor_2d>::execute(ex, hello_world(), shape);

  return 0;
}

