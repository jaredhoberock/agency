#include "hip/hip_runtime.h"
#include <agency/agency.hpp>
#include <agency/execution/executor/detail/utility.hpp>
#include <agency/cuda.hpp>
#include <cassert>
#include <iostream>

#include "../../test_executors.hpp"

template<class Executor>
void test(Executor exec)
{
  std::atomic<int> counter{0};

  using shape_type = agency::executor_shape_t<Executor>;
  using index_type = agency::executor_index_t<Executor>;

  shape_type shape{10};
  
  agency::detail::bulk_sync_execute_with_void_result(exec,
    [&](index_type idx, int& shared_arg)
    {
      counter += shared_arg;
    },
    shape,
    []{ return 13; } // shared_arg
  );
  
  assert(counter == 13 * 10);
}


__managed__ int increment_me;

template<class Executor>
void test2(Executor exec)
{
  using shape_type = agency::executor_shape_t<Executor>;
  using index_type = agency::executor_index_t<Executor>;

  shape_type shape{10,10};

  increment_me = 0;
  
  agency::detail::bulk_sync_execute_with_void_result(exec,
    [] __device__ (index_type idx, int& outer_arg, int& inner_arg)
    {
      atomicAdd(&increment_me, outer_arg + inner_arg);
    },
    shape,
    [] __host__ __device__ { return 7; }, // outer_arg
    [] __host__ __device__ { return 13; } // inner_arg
  );
  
  assert(increment_me == (7 + 13) * 10 * 10);
}


int main()
{
  test(bulk_synchronous_executor());
  test(bulk_asynchronous_executor());
  test(bulk_continuation_executor());
  test(not_a_bulk_synchronous_executor());
  test(not_a_bulk_asynchronous_executor());
  test(not_a_bulk_continuation_executor());
  test(complete_bulk_executor());

  test2(agency::cuda::grid_executor());

  std::cout << "OK" << std::endl;

  return 0;
}

