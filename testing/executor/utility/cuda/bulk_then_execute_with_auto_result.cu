#include "hip/hip_runtime.h"
#include <agency/agency.hpp>
#include <agency/execution/executor/detail/utility.hpp>
#include <agency/cuda.hpp>
#include <iostream>

#include "../../test_executors.hpp"


__managed__ int increment_me;


template<class Executor>
void test_with_void_predecessor_returning_void(Executor exec)
{
  agency::executor_shape_t<Executor> shape{100};

  auto predecessor_future = agency::make_ready_future<void>(exec);
  
  int shared_arg = 0;
  
  int increment_me = 0;
  std::mutex mut;
  auto fut = agency::detail::bulk_then_execute_with_auto_result(exec, [&](size_t idx, int& shared_arg)
  {
    mut.lock();
    increment_me += 1;
    ++shared_arg;
    mut.unlock();
  },
  shape,
  predecessor_future,
  [&]
  {
    return std::ref(shared_arg);
  });
  
  fut.wait();
  
  assert(increment_me == shape);
  assert(shared_arg == shape);
}


template<class Executor>
void test_with_void_predecessor_returning_results(Executor exec)
{
  auto predecessor_future = agency::make_ready_future<void>(exec);

  using shape_type = agency::executor_shape_t<Executor>;
  using index_type = agency::executor_index_t<Executor>;

  size_t shape = 10;
  
  auto f = agency::detail::bulk_then_execute_with_auto_result(exec,
    [](index_type idx, std::vector<int>& shared_arg)
    {
      return shared_arg[idx];
    },
    shape,
    predecessor_future,
    [=]{ return std::vector<int>(shape, 13); }  // shared_arg
  );
  
  auto result = f.get();
  
  using container_type = agency::executor_container_t<Executor,int>;
  assert(container_type(shape, 13) == result);
}


template<class Executor>
void test_with_non_void_predecessor_returning_void(Executor exec)
{
  agency::executor_shape_t<Executor> shape{100};

  auto predecessor_future = agency::make_ready_future<int>(exec, 13);
  
  int shared_arg = 0;
  
  int increment_me = 0;
  std::mutex mut;
  auto fut = agency::detail::bulk_then_execute_with_auto_result(exec, [&](size_t idx, int& predecessor, int& shared_arg)
  {
    mut.lock();
    increment_me += predecessor;
    ++shared_arg;
    mut.unlock();
  },
  shape,
  predecessor_future,
  [&]
  {
    return std::ref(shared_arg);
  });
  
  fut.wait();
  
  assert(increment_me == shape * 13);
  assert(shared_arg == shape);
}


template<class Executor>
void test_with_non_void_predecessor_returning_results(Executor exec)
{
  auto predecessor_future = agency::make_ready_future<int>(exec, 7);

  using shape_type = agency::executor_shape_t<Executor>;
  using index_type = agency::executor_index_t<Executor>;

  size_t shape = 10;
  
  auto f = agency::detail::bulk_then_execute_with_auto_result(exec,
    [](index_type idx, int& predecessor, std::vector<int>& shared_arg)
    {
      return predecessor + shared_arg[idx];
    },
    shape,
    predecessor_future,
    [=]{ return std::vector<int>(shape, 13); }  // shared_arg
  );
  
  auto result = f.get();
  
  using container_type = agency::executor_container_t<Executor,int>;
  assert(container_type(shape, 7 + 13) == result);
}


template<class Executor>
void test_with_void_predecessor_returning_void2(Executor exec)
{
  agency::executor_shape_t<Executor> shape{10,10};

  auto predecessor_future = agency::make_ready_future<void>(exec);

  increment_me = 0;

  using index_type = agency::executor_index_t<Executor>;
  
  auto fut = agency::detail::bulk_then_execute_with_auto_result(exec, [] __device__ (index_type idx, int& outer_shared_arg, int& inner_shared_arg)
  {
    atomicAdd(&increment_me, outer_shared_arg + inner_shared_arg);
  },
  shape,
  predecessor_future,
  [] __host__ __device__ { return 7; },
  [] __host__ __device__ { return 13; }
  );
  
  fut.wait();
  
  assert(increment_me == shape[0] * shape[1] * (7 + 13));
}


template<class Executor>
void test_with_void_predecessor_returning_results2(Executor exec)
{
  auto predecessor_future = agency::make_ready_future<void>(exec);

  using shape_type = agency::executor_shape_t<Executor>;
  using index_type = agency::executor_index_t<Executor>;

  shape_type shape{10,10};
  
  auto f = agency::detail::bulk_then_execute_with_auto_result(exec,
    [] __host__ __device__ (index_type idx, int& outer_shared_arg, int& inner_shared_arg)
    {
      return outer_shared_arg + inner_shared_arg;
    },
    shape,
    predecessor_future,
    [] __host__ __device__ { return 7; }, // outer_shared_arg
    [] __host__ __device__ { return 13; }   // inner_shared_arg
  );
  
  auto result = f.get();
  
  using container_type = agency::executor_container_t<Executor,int>;
  assert(container_type(shape, 7 + 13) == result);
}


template<class Executor>
void test_with_non_void_predecessor_returning_void2(Executor exec)
{
  agency::executor_shape_t<Executor> shape{10,10};

  auto predecessor_future = agency::make_ready_future<int>(exec, 42);

  increment_me = 0;

  using index_type = agency::executor_index_t<Executor>;
  
  auto fut = agency::detail::bulk_then_execute_with_auto_result(exec, [] __device__ (index_type idx, int& predecessor, int& outer_shared_arg, int& inner_shared_arg)
  {
    atomicAdd(&increment_me, predecessor + outer_shared_arg + inner_shared_arg);
  },
  shape,
  predecessor_future,
  [] __host__ __device__ { return 7; },
  [] __host__ __device__ { return 13; }
  );
  
  fut.wait();
  
  assert(increment_me == shape[0] * shape[1] * (42 + 7 + 13));
}


template<class Executor>
void test_with_non_void_predecessor_returning_results2(Executor exec)
{
  agency::executor_shape_t<Executor> shape{10,10};

  auto predecessor_future = agency::make_ready_future<int>(exec, 42);

  using index_type = agency::executor_index_t<Executor>;
  
  auto fut = agency::detail::bulk_then_execute_with_auto_result(exec, [] __host__ __device__ (index_type idx, int& predecessor, int& outer_shared_arg, int& inner_shared_arg)
  {
    return predecessor + outer_shared_arg + inner_shared_arg;
  },
  shape,
  predecessor_future,
  [] __host__ __device__ { return 7; },
  [] __host__ __device__ { return 13; }
  );
  
  auto result = fut.get();
  
  using container_type = agency::executor_container_t<Executor,int>;
  assert(container_type(shape, 42 + 7 + 13) == result);
}


int main()
{
  test_with_void_predecessor_returning_void(bulk_synchronous_executor());
  test_with_void_predecessor_returning_void(bulk_asynchronous_executor());
  test_with_void_predecessor_returning_void(bulk_continuation_executor());
  test_with_void_predecessor_returning_void(not_a_bulk_synchronous_executor());
  test_with_void_predecessor_returning_void(not_a_bulk_asynchronous_executor());
  test_with_void_predecessor_returning_void(not_a_bulk_continuation_executor());
  test_with_void_predecessor_returning_void(complete_bulk_executor());

  test_with_void_predecessor_returning_results(bulk_synchronous_executor());
  test_with_void_predecessor_returning_results(bulk_asynchronous_executor());
  test_with_void_predecessor_returning_results(bulk_continuation_executor());
  test_with_void_predecessor_returning_results(not_a_bulk_synchronous_executor());
  test_with_void_predecessor_returning_results(not_a_bulk_asynchronous_executor());
  test_with_void_predecessor_returning_results(not_a_bulk_continuation_executor());
  test_with_void_predecessor_returning_results(complete_bulk_executor());

  test_with_non_void_predecessor_returning_void(bulk_synchronous_executor());
  test_with_non_void_predecessor_returning_void(bulk_asynchronous_executor());
  test_with_non_void_predecessor_returning_void(bulk_continuation_executor());
  test_with_non_void_predecessor_returning_void(not_a_bulk_synchronous_executor());
  test_with_non_void_predecessor_returning_void(not_a_bulk_asynchronous_executor());
  test_with_non_void_predecessor_returning_void(not_a_bulk_continuation_executor());
  test_with_non_void_predecessor_returning_void(complete_bulk_executor());

  test_with_non_void_predecessor_returning_results(bulk_synchronous_executor());
  test_with_non_void_predecessor_returning_results(bulk_asynchronous_executor());
  test_with_non_void_predecessor_returning_results(bulk_continuation_executor());
  test_with_non_void_predecessor_returning_results(not_a_bulk_synchronous_executor());
  test_with_non_void_predecessor_returning_results(not_a_bulk_asynchronous_executor());
  test_with_non_void_predecessor_returning_results(not_a_bulk_continuation_executor());
  test_with_non_void_predecessor_returning_results(complete_bulk_executor());


  test_with_void_predecessor_returning_void2(agency::cuda::grid_executor());
  test_with_void_predecessor_returning_results2(agency::cuda::grid_executor());
  test_with_non_void_predecessor_returning_void2(agency::cuda::grid_executor());
  test_with_non_void_predecessor_returning_results2(agency::cuda::grid_executor());

  std::cout << "OK" << std::endl;
  
  return 0;
}

