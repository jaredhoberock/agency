#include <agency/agency.hpp>
#include <agency/execution/executor/detail/utility.hpp>
#include <agency/cuda.hpp>
#include <iostream>

#include "../../test_executors.hpp"

template<class Executor>
void test(Executor exec)
{
  using shape_type = agency::executor_shape_t<Executor>;
  using index_type = agency::executor_index_t<Executor>;

  size_t shape = 10;
  
  auto f = agency::detail::bulk_async_execute_with_collected_result(exec,
    [](index_type idx, std::vector<int>& shared_arg)
    {
      return shared_arg[idx];
    },
    shape,
    [=]{ return std::vector<int>(shape); },    // results
    [=]{ return std::vector<int>(shape, 13); } // shared_arg
  );

  auto result = f.get();
  
  assert(std::vector<int>(shape, 13) == result);
}


template<class Executor>
void test2(Executor exec)
{
  using shape_type = agency::executor_shape_t<Executor>;
  using index_type = agency::executor_index_t<Executor>;

  shape_type shape{10,10};

  using container_type = agency::executor_container_t<Executor,int>;
  
  auto f = agency::detail::bulk_async_execute_with_collected_result(exec,
    [] __host__ __device__ (index_type idx, int& outer_shared_arg, int& inner_shared_arg)
    {
      return outer_shared_arg + inner_shared_arg;
    },
    shape,
    [=] __host__ __device__ { return container_type(shape); }, // results
    [] __host__ __device__  { return 7; },                     // outer_shared_arg
    [] __host__ __device__  { return 13; }                     // inner_shared_arg
  );

  auto result = f.get();
  
  assert(container_type(shape, 7 + 13) == result);
}


int main()
{
  test(bulk_synchronous_executor());
  test(bulk_asynchronous_executor());
  test(bulk_continuation_executor());
  test(not_a_bulk_synchronous_executor());
  test(not_a_bulk_asynchronous_executor());
  test(not_a_bulk_continuation_executor());
  test(complete_bulk_executor());

  test2(agency::cuda::grid_executor());

  std::cout << "OK" << std::endl;

  return 0;
}

