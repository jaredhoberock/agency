#include "hip/hip_runtime.h"
#include <agency/bulk_async.hpp>
#include <agency/cuda/execution_policy.hpp>
#include <atomic>

__managed__ int counter;

template<class ExecutionPolicy>
void test(ExecutionPolicy policy)
{
  using agent = typename ExecutionPolicy::execution_agent_type;
  using agent_traits = agency::execution_agent_traits<agent>;

  {
    // bulk_invoke with no parameters

    counter = 0;

    auto f = agency::bulk_async(policy, [] __host__ __device__ (agent& self)
    {
#ifdef __CUDA_ARCH__
      atomicAdd(&counter, 1);
#endif
    });

    f.wait();

    size_t num_agents = agent_traits::domain(policy.param()).size();

    assert(counter == num_agents);
  }

  {
    // bulk_invoke with one parameter

    int val = 13;

    counter = 0;

    auto f = agency::bulk_async(policy,
      [] __host__ __device__ (agent& self, int val)
      {
#ifdef __CUDA_ARCH__
        atomicAdd(&counter, val);
#endif
      },
      val
    );

    f.wait();

    size_t num_agents = agent_traits::domain(policy.param()).size();

    assert(counter == num_agents * 13);
  }

  {
    // bulk_invoke with one shared parameter

    int val = 13;

    counter = 0;

    auto f = agency::bulk_async(policy,
      [] __host__ __device__ (agent& self, int& val)
      {
#ifdef __CUDA_ARCH__
        atomicAdd(&counter, val);
#endif
      },
      agency::share(val)
    );

    f.wait();

    size_t num_agents = agent_traits::domain(policy.param()).size();

    assert(counter == num_agents * 13);
  }
}

int main()
{
  using namespace agency::cuda;

  test(con(10));
  test(par(10));

  test(par(10, con(10)));

  std::cout << "OK" << std::endl;

  return 0;
}

