#include "hip/hip_runtime.h"
#include <agency/agency.hpp>
#include <agency/cuda.hpp>
#include <cassert>
#include <iostream>
#include <atomic>


template<int init>
class initialized_to
{
  public:
    __host__ __device__
    initialized_to()
      : value_(init)
    {}

    __host__ __device__
    int& value()
    {
      return value_;
    }

    __host__ __device__
    const int& value() const
    {
      return value_;
    }

  private:
    int value_;
};



std::atomic<int> h_increment_me(0);
__managed__ int  d_increment_me;


template<class ExecutionPolicy>
void test(ExecutionPolicy policy)
{
  {
    // bulk_invoke with one automatic shared parameter returning results

    auto result = agency::bulk_invoke(policy(10),
      [] __host__ __device__ (typename ExecutionPolicy::execution_agent_type& self, initialized_to<13>& shared)
    {
      return shared.value();
    });

    using executor_type = typename ExecutionPolicy::executor_type;
    using container_type = agency::executor_container_t<executor_type,int>;

    assert(container_type(10,13) == result);
  }

  {
    // bulk_invoke with two automatic shared parameters returning results

    auto result = agency::bulk_invoke(policy(10),
      [] __host__ __device__ (typename ExecutionPolicy::execution_agent_type& self, initialized_to<13>& shared1, initialized_to<7>& shared2)
    {
      return shared1.value() + shared2.value();
    });

    using executor_type = typename ExecutionPolicy::executor_type;
    using container_type = agency::executor_container_t<executor_type,int>;

    assert(container_type(10,13 + 7) == result);
  }

  {
    // bulk_invoke with one automatic shared parameter returning void

    h_increment_me = 0;
    d_increment_me = 0;

    agency::bulk_invoke(policy(10),
      [] __host__ __device__ (typename ExecutionPolicy::execution_agent_type& self, initialized_to<13>& val)
    {
#ifdef __CUDA_ARCH__
      atomicAdd(&d_increment_me, val.value());
#else
      h_increment_me += val.value();
#endif
    });

    assert((h_increment_me == 13 * 10) || (d_increment_me == 13 * 10));
  }

  {
    // bulk_invoke with two automatic shared parameters returning void

    h_increment_me = 0;
    d_increment_me = 0;

    agency::bulk_invoke(policy(10),
      [] __host__ __device__ (typename ExecutionPolicy::execution_agent_type& self, initialized_to<13>& shared1, initialized_to<7>& shared2)
    {
#ifdef __CUDA_ARCH__
      atomicAdd(&d_increment_me, shared1.value() + shared2.value());
#else
      h_increment_me += shared1.value() + shared2.value();
#endif
    });

    assert((h_increment_me == (13 + 7) * 10) || (d_increment_me == (13 + 7) * 10));
  }
}


int main()
{
  test(agency::seq);
  test(agency::con);
  test(agency::par);
  test(agency::unseq);
  test(agency::cuda::par);

  std::cout << "OK" << std::endl;

  return 0;
}

