#include "hip/hip_runtime.h"
#include <agency/agency.hpp>
#include <agency/cuda.hpp>
#include <atomic>

__managed__ int counter;

template<class ExecutionPolicy>
void test(ExecutionPolicy policy)
{
  using agent = typename ExecutionPolicy::execution_agent_type;
  using agent_traits = agency::execution_agent_traits<agent>;

  {
    // bulk_invoke with no parameters

    counter = 0;

    auto f = agency::bulk_async(policy, [] __host__ __device__ (agent&)
    {
#ifdef __CUDA_ARCH__
      atomicAdd(&counter, 1);
#endif
    });

    f.wait();

    int num_agents = agent_traits::domain(policy.param()).size();

    assert(counter == num_agents);
  }

  {
    // bulk_invoke with one parameter

    int val = 13;

    counter = 0;

    auto f = agency::bulk_async(policy,
      [] __host__ __device__ (agent&, int val)
      {
        // WAR unused parameter warning
        (void)val;

#ifdef __CUDA_ARCH__
        atomicAdd(&counter, val);
#endif
      },
      val
    );

    f.wait();

    int num_agents = agent_traits::domain(policy.param()).size();

    assert(counter == num_agents * 13);
  }

  {
    // bulk_invoke with one shared parameter

    int val = 13;

    counter = 0;

    auto f = agency::bulk_async(policy,
      [] __host__ __device__ (agent&, int& val)
      {
        // WAR unused parameter warning
        (void)val;

#ifdef __CUDA_ARCH__
        atomicAdd(&counter, val);
#endif
      },
      agency::share(val)
    );

    f.wait();

    int num_agents = agent_traits::domain(policy.param()).size();

    assert(counter == num_agents * 13);
  }
}

int main()
{
  using namespace agency::cuda;

  test(con(10));
  test(par(10));

  test(par(10, con(10)));

  std::cout << "OK" << std::endl;

  return 0;
}

