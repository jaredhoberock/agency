#include "hip/hip_runtime.h"
#include "utility.hpp"

#include <agency/agency.hpp>
#include <agency/cuda.hpp>

#include <iostream>
#include <vector>
#include <cassert>

void sequential_csr_spmv(int num_rows,
                         const int* row_offsets,
                         const int* column_indices,
                         const float* values,
                         const float* x,
                         float *y)
{
  for(int row = 0; row < num_rows; ++row)
  {
    float dot = 0;
    
    int row_start = row_offsets[row];
    int row_end = row_offsets[row + 1];

    for(int value_idx = row_start; value_idx < row_end; ++value_idx)
    {
      dot += values[value_idx] * x[column_indices[value_idx]];
    }

    y[row] += dot;
  }
}

void parallel_csr_spmv(int num_rows,
                       const int* row_offsets,
                       const int* column_indices,
                       const float* values,
                       const float* x,
                       float *y)
{
  constexpr int block_size = 128;
  constexpr int warp_size = 32;

  // choose the number of blocks
  constexpr int warps_per_block = block_size / warp_size;
  constexpr int max_threads = 30 * 1024;
  constexpr int max_blocks = max_threads / block_size;
  int num_blocks = (num_rows + warps_per_block - 1) / warps_per_block;
  num_blocks = std::min(max_blocks, num_blocks);

  using namespace agency;

  bulk_invoke(cuda::grid(num_blocks, block_size), [=] __device__ (cuda::grid_agent& self)
  {
    __shared__ volatile float sdata[warps_per_block * warp_size + warp_size/2];      // padded to avoid reduction ifs
    __shared__ volatile int ptrs[warps_per_block][2];
    
    const int thread_id   = block_size * blockIdx.x + threadIdx.x;  // global thread index
    const int thread_lane = threadIdx.x & (warp_size-1);            // thread index within the warp
    const int warp_id     = thread_id   / warp_size;                // global warp index
    const int warp_lane   = threadIdx.x / warp_size;                // warp index within the CTA
    const int num_warps   = warps_per_block * gridDim.x;   // total number of active warps

    for(int row = warp_id; row < num_rows; row += num_warps)
    {
      // use two threads to fetch row_offsets[row] and row_offsets[row+1]
      // this is considerably faster than the straightforward version
      if(thread_lane < 2)
      {
        ptrs[warp_lane][thread_lane] = row_offsets[row + thread_lane];
      }

      const int row_start = ptrs[warp_lane][0];                   //same as: row_start = Ap[row];
      const int row_end   = ptrs[warp_lane][1];                   //same as: row_end   = Ap[row+1];

      // compute local sum
      float sum = 0;
      for(int jj = row_start + thread_lane; jj < row_end; jj += warp_size)
      {
        sum += values[jj] * x[column_indices[jj]];
      }

      // reduce local sums to row sum (ASSUME: warpsize 32)
      sdata[threadIdx.x] = sum;
      sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; 
      sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8];
      sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4];
      sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2];
      sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1];

      // first thread writes warp result
      if(thread_lane == 0)
      {
        y[row] += sdata[threadIdx.x];
      }
    } // end for row
  });
}


int main()
{
  // create shorthand for the vectors we will use in this program
  using index_vector = std::vector<int, agency::cuda::allocator<int>>;
  using value_vector = std::vector<float, agency::cuda::allocator<float>>;

  {
    // test our implementation's correctness on simple SPMV problem
    index_vector row_offsets;
    index_vector column_indices;
    value_vector values;

    value_vector x, reference;

    create_simple_csr_spmv_problem(row_offsets, column_indices, values, x, reference);
    int num_rows = row_offsets.size() - 1;

    value_vector y(num_rows);

    parallel_csr_spmv(num_rows, row_offsets.data(), column_indices.data(), values.data(), x.data(), y.data());

    std::cout << "y: [";
    for(int row = 0; row < y.size(); ++row)
    {
      std::cout << " " << y[row];
    }
    std::cout << " ]" << std::endl;

    assert(almost_equal(reference, y));
  }

  {
    index_vector row_offsets;
    index_vector column_indices;
    value_vector values;

    // create a CSR matrix
    int num_rows = 128;
    int num_columns = num_rows;
    laplacian_5pt(num_rows, row_offsets, column_indices, values);

    // XXX should generate random matrix values

    // XXX should generate random vector values
    value_vector x(num_columns, 1);

    // compute reference solution
    value_vector reference(num_rows);
    sequential_csr_spmv(num_rows, row_offsets.data(), column_indices.data(), values.data(), x.data(), reference.data());

    // multiply
    value_vector y(num_rows);
    parallel_csr_spmv(num_rows, row_offsets.data(), column_indices.data(), values.data(), x.data(), y.data());

    assert(almost_equal(reference, y));
  }

  std::cout << "OK" << std::endl;

  return 0;
}

